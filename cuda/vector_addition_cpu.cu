#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vecAddKernel(float *A_h, float *B_h, float *C_h, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n) {
        C_h[i] = A_h[i] + B_h[i];
    }

}

int main() {
    int N = 100;

    // Allocating floats in CPU memory
    float *A_h = (float *)malloc(N * sizeof(float));
    float *B_h = (float *)malloc(N * sizeof(float));
    float *C_h = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        A_h[i] = i*1;
        B_h[i] = i*2;
    }

    float *A_d, *B_d, *C_d;

    float size = N*sizeof(float);

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    int gridsize(1);
    int blocksize(100);

    vecAddKernel<<<gridsize, blocksize>>>(A_d, B_d, C_d, N);

    hipMemcpy(C_h, C_d, N*sizeof(float), hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(error));
    }

    printf("here\n\n");

    for (int i = 0; i < N; i++) {
        printf("%0.1f\n", C_h[i]);
    }

    free(A_h);
    free(B_h);
    free(C_h);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;
}
